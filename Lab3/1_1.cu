#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define G 6.67E-11

__global__ void calculate_force_cuda(float *masses, float *array_x, float *array_y, 
                                     float *fx, float *fy, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n) {
        float local_fx = 0.0f;
        float local_fy = 0.0f;

        for (int j = 0; j < n; ++j) {
            if (i != j) {
                float dx = array_x[j] - array_x[i];
                float dy = array_y[j] - array_y[i];
                float squared_dist = dx * dx + dy * dy + 1e-9f; // Avoid division by zero
                float dist = sqrtf(squared_dist);
                float force = G * masses[i] * masses[j] / (squared_dist * dist);
                local_fx += force * dx;
                local_fy += force * dy;
            }
        }

        fx[i] = local_fx;
        fy[i] = local_fy;
    }
}

// CUDA kernel for updating positions and velocities of bodies
__global__ void update_points_cuda(float *fx, float *fy, float *masses, float *array_x, 
                                    float *array_y, float *v_x, float *v_y, int n, float delta_t) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n) {
        v_x[i] += (fx[i] / masses[i]) * delta_t;
        v_y[i] += (fy[i] / masses[i]) * delta_t;
        array_x[i] += v_x[i] * delta_t;
        array_y[i] += v_y[i] * delta_t;
    }
}

// Host function for generating initial body data
void generate_bodies(float *masses, float *array_x, float *array_y, float *v_x, float *v_y, int n) {
    for (int i = 0; i < n; ++i) {
        masses[i] = ((float)rand()) / (RAND_MAX >> 10);
        array_x[i] = 2.0 * ((float)rand()) / RAND_MAX - 1.0;
        array_y[i] = 2.0 * ((float)rand()) / RAND_MAX - 1.0;
        v_x[i] = 2.0 * ((float)rand()) / RAND_MAX - 1.0;
        v_y[i] = 2.0 * ((float)rand()) / RAND_MAX - 1.0;
    }
}

int main(int argc, char *argv[]) {
    int n;
    float t_end;
    n = atoi(argv[1]);
    t_end = atof(argv[2]);
    float delta_t = t_end / 100.0;

    // Host memory allocation
    float *masses = (float *)malloc(n * sizeof(float));
    float *array_x = (float *)malloc(n * sizeof(float));
    float *array_y = (float *)malloc(n * sizeof(float));
    float *v_x = (float *)malloc(n * sizeof(float));
    float *v_y = (float *)malloc(n * sizeof(float));
    float *fx = (float *)malloc(n * sizeof(float));
    float *fy = (float *)malloc(n * sizeof(float));

    generate_bodies(masses, array_x, array_y, v_x, v_y, n);

    // Device memory allocation
    float *d_masses, *d_array_x, *d_array_y, *d_v_x, *d_v_y, *d_fx, *d_fy;
    hipMalloc((void **)&d_masses, n * sizeof(float));
    hipMalloc((void **)&d_array_x, n * sizeof(float));
    hipMalloc((void **)&d_array_y, n * sizeof(float));
    hipMalloc((void **)&d_v_x, n * sizeof(float));
    hipMalloc((void **)&d_v_y, n * sizeof(float));
    hipMalloc((void **)&d_fx, n * sizeof(float));
    hipMalloc((void **)&d_fy, n * sizeof(float));

    // Copy data from host to device
    hipMemcpy(d_masses, masses, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_array_x, array_x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_array_y, array_y, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_v_x, v_x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_v_y, v_y, n * sizeof(float), hipMemcpyHostToDevice);

    // Determine CUDA grid and block sizes
    int threads_per_block = 256;
    int blocks_per_grid = (n + threads_per_block - 1) / threads_per_block;

    float current_time = 0.0;
    while (current_time < t_end) {
        // Print current state
        printf("%f ", current_time);
        for (int i = 0; i < n; ++i) {
            printf("%f %f ", array_x[i], array_y[i]);
        }
        printf("\n");

        // Calculate forces on the device
        calculate_force_cuda<<<blocks_per_grid, threads_per_block>>>(d_masses, d_array_x, d_array_y, d_fx, d_fy, n);

        // Update positions and velocities on the device
        update_points_cuda<<<blocks_per_grid, threads_per_block>>>(d_fx, d_fy, d_masses, d_array_x, d_array_y, d_v_x, d_v_y, n, delta_t);

        // Copy updated positions back to host
        hipMemcpy(array_x, d_array_x, n * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(array_y, d_array_y, n * sizeof(float), hipMemcpyDeviceToHost);

        current_time += delta_t;
    }

    // Free device memory
    hipFree(d_masses);
    hipFree(d_array_x);
    hipFree(d_array_y);
    hipFree(d_v_x);
    hipFree(d_v_y);
    hipFree(d_fx);
    hipFree(d_fy);

    // Free host memory
    free(masses);
    free(array_x);
    free(array_y);
    free(v_x);
    free(v_y);
    free(fx);
    free(fy);

    return 0;
}
